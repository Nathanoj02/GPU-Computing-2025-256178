#include "hip/hip_runtime.h"
#include "data_cuda.cuh"

#include <malloc.h>

#define MAX_LINE 256

#define EPS 1

void set_array_random(float *arr, int size, float max_value);


int read_from_file(char *path, DataCuda *data)
{
    FILE *fp = fopen(path, "r");

    // Check if file is opened
    if (!fp)
    {
        perror("File opening failed!\n");
        return -1;
    }

    char line[MAX_LINE];

    // Skip first comment lines
    while (fgets(line, MAX_LINE, fp)) 
    {
        if (line[0] != '%')
            break;
    }

    // Get #rows, #cols, #vals
    int rows, cols, vals;
    if (sscanf(line, "%d %d %d", &rows, &cols, &vals) != 3)
    {
        fclose(fp);
        printf("Error: Invalid matrix header format\n");
        return -2;
    }

    data->row_num = rows;
    data->col_num = cols;
    data->val_num = vals;

    int counter = 0;

    // Alloc memory
    data->row = (int *) malloc(sizeof(int) * data->val_num);
    data->col = (int *) malloc(sizeof(int) * data->val_num);
    data->val = (float *) malloc(sizeof(float) * data->val_num);
    
    // Read matrix entries
    int row, col;
    float value;
    while (fscanf(fp, "%d %d %f", &row, &col, &value) == 3)
    {
        data->row[counter] = row - 1;    // In data, indexing starts at 1
        data->col[counter] = col - 1;
        data->val[counter] = value;
        
        counter++;
    }
    
    fclose(fp);

    data->arr = (float *) malloc(sizeof(float) * data->col_num);
    data->res = (float *) malloc(sizeof(float) * data->row_num);

    // Random array for multiplication
    set_array_random(data->arr, data->col_num, 1);

    return 0;
}

/**
 * @brief Fills array with random values in [0, max_value]
 */
void set_array_random(float *arr, int size, float max_value)
{
    srand((unsigned) time(NULL));

    for (int i = 0; i < size; i++) {
        arr[i] = rand() / (float) RAND_MAX * max_value;
    }
}

/**
 * @details Result should be [8, 15, 0, 2, 18, 10]
 */
DataCuda test_data()
{
    DataCuda data;

    data.row_num = 6;
    data.col_num = 4;
    data.val_num = 7;

    data.row = (int *) malloc(sizeof(int) * data.val_num);
    data.col = (int *) malloc(sizeof(int) * data.val_num);
    data.val = (float *) malloc(sizeof(float) * data.val_num);
    data.arr = (float *) malloc(sizeof(float) * data.col_num);
    data.res = (float *) calloc(data.row_num, sizeof(float));

    int row[] = {0, 0, 3, 1, 4, 5, 5};
    int col[] = {1, 2, 0, 3, 1, 0, 2};
    float val[] = {1, 2, 1, 5, 3, 4, 2};
    float arr[] = {2, 6, 1, 3};

    for (int i = 0; i < data.val_num; i++) {
        data.row[i] = row[i];
        data.col[i] = col[i];
        data.val[i] = val[i];
    }
    for (int i = 0; i < data.col_num; i++) {
        data.arr[i] = arr[i];
    }

    return data;
}


// Merges two subarrays of row[], while maintaining col[] and val[]
void merge(int *row, int *col, float *val, int left, int mid, int right) {
    int n1 = mid - left + 1;
    int n2 = right - mid;

    // Temporary arrays for row, col, and val
    int *leftRow = (int *)malloc(n1 * sizeof(int));
    int *leftCol = (int *)malloc(n1 * sizeof(int));
    float *leftVal = (float *)malloc(n1 * sizeof(float));

    int *rightRow = (int *)malloc(n2 * sizeof(int));
    int *rightCol = (int *)malloc(n2 * sizeof(int));
    float *rightVal = (float *)malloc(n2 * sizeof(float));

    if (!leftRow || !rightRow || !leftCol || !rightCol || !leftVal || !rightVal) {
        printf("Memory allocation failed!\n");
        exit(1);
    }

    // Copy data to temporary arrays
    for (int i = 0; i < n1; i++) {
        leftRow[i] = row[left + i];
        leftCol[i] = col[left + i];
        leftVal[i] = val[left + i];
    }
    for (int j = 0; j < n2; j++) {
        rightRow[j] = row[mid + 1 + j];
        rightCol[j] = col[mid + 1 + j];
        rightVal[j] = val[mid + 1 + j];
    }

    // Merge the temporary arrays back into row[], col[], and val[]
    int i = 0, j = 0, k = left;
    while (i < n1 && j < n2) {
        if (leftRow[i] <= rightRow[j]) {
            row[k] = leftRow[i];
            col[k] = leftCol[i];
            val[k] = leftVal[i];
            i++;
        } else {
            row[k] = rightRow[j];
            col[k] = rightCol[j];
            val[k] = rightVal[j];
            j++;
        }
        k++;
    }

    // Copy the remaining elements of left arrays, if any
    while (i < n1) {
        row[k] = leftRow[i];
        col[k] = leftCol[i];
        val[k] = leftVal[i];
        i++;
        k++;
    }

    // Copy the remaining elements of right arrays, if any
    while (j < n2) {
        row[k] = rightRow[j];
        col[k] = rightCol[j];
        val[k] = rightVal[j];
        j++;
        k++;
    }

    // Free allocated memory
    free(leftRow);
    free(rightRow);
    free(leftCol);
    free(rightCol);
    free(leftVal);
    free(rightVal);
}

// The subarray to be sorted is in the index range [left-right]
void mergeSort(int *row, int *col, float *val, int left, int right) {
    if (left < right) {
      
        // Calculate the midpoint
        int mid = left + (right - left) / 2;

        // Sort first and second halves
        mergeSort(row, col, val, left, mid);
        mergeSort(row, col, val, mid + 1, right);

        // Merge the sorted halves
        merge(row, col, val, left, mid, right);
    }
}


bool check_data(float *check, float *base, int size)
{
    for (int i = 0; i < size; i++)
    {
        if (abs(check[i] - base[i]) > EPS) {
            printf("\nRiga %d:\t", i);
            printf("%lf vs %lf\n\n", check[i], base[i]);
            return false;
        }
    }
    
    return true;
}