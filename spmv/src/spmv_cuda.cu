#include "hip/hip_runtime.h"
#include "spmv_cuda.cuh"
#include "error.cuh"
#include "utils.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <malloc.h>
#include <math.h>

#include <sys/time.h>

SmdvInfoOrdered init_mul (size_t num_rows, size_t num_cols, size_t val_num);
void exec_mul (
    float *dst, size_t *col, float *val, float *arr, size_t *thread_start,
    size_t num_rows, size_t num_cols, size_t val_num, SmdvInfoOrdered& smdv_info
);
void deinit_mul (SmdvInfoOrdered &smdv_info);


__global__
void mul_kernel (
    float *dst, size_t *col, float *val, float *arr, size_t *thread_start,
    size_t num_rows, size_t val_num
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread is in range
    if (idx >= num_rows)
        return;

    float p = 0;

    int thread_end = thread_start[idx + 1];
    for (int i = thread_start[idx]; i < thread_end; i++)
    {
        p += val[i] * arr[col[i]];
    }

    dst[idx] = p;
}


void mul_cuda_ordered (
    float *dst, size_t *col, float *val, float *arr, size_t *thread_start,
    size_t num_rows, size_t num_cols, size_t val_num
)
{
    // struct timeval t1 = {0, 0}, t2 = {0, 0};
    // gettimeofday(&t1, (struct timezone *) 0);

    auto smdv_info = init_mul(num_rows, num_cols, val_num);

    // gettimeofday(&t2, (struct timezone *) 0);

    // double time1 = ((t2.tv_sec - t1.tv_sec) * 1.e6 + (t2.tv_usec - t1.tv_usec));

    // printf("\nInit: %lf usec\n", time1);
    
    // gettimeofday(&t1, (struct timezone *) 0);
    
    
    exec_mul(dst, col, val, arr, thread_start, num_rows, num_cols, val_num, smdv_info);
    
    // gettimeofday(&t2, (struct timezone *) 0);
    
    // double time2 = ((t2.tv_sec - t1.tv_sec) * 1.e6 + (t2.tv_usec - t1.tv_usec));
    // printf("Exec: %lf usec\n", time2);
    
    // gettimeofday(&t1, (struct timezone *) 0);
    deinit_mul(smdv_info);
    // gettimeofday(&t2, (struct timezone *) 0);
    // double time3 = ((t2.tv_sec - t1.tv_sec) * 1.e6 + (t2.tv_usec - t1.tv_usec));
    // printf("End: %lf usec\n", time3);
}


void exec_mul (
    float *dst, size_t *col, float *val, float *arr, size_t *thread_start,
    size_t num_rows, size_t num_cols, size_t val_num, SmdvInfoOrdered& smdv_info
)
{
    // Memcpy
    SAFE_CALL( hipMemcpy(smdv_info.d_val, val, sizeof(float) * val_num, hipMemcpyHostToDevice) );
    SAFE_CALL( hipMemcpy(smdv_info.d_arr, arr, sizeof(float) * num_cols, hipMemcpyHostToDevice) );
    SAFE_CALL( hipMemcpy(smdv_info.d_col, col, sizeof(size_t) * val_num, hipMemcpyHostToDevice) );
    SAFE_CALL( hipMemcpy(smdv_info.d_thread_start, thread_start, sizeof(size_t) * (num_rows + 1), hipMemcpyHostToDevice) );

    dim3 dim_grid = dim3(smdv_info.dim.grid.x, smdv_info.dim.grid.y, smdv_info.dim.grid.z);
    dim3 dim_block = dim3(smdv_info.dim.block.x, smdv_info.dim.block.y, smdv_info.dim.block.z);

    // Kernel invocation
    mul_kernel <<< dim_grid, dim_block >>> (
        smdv_info.d_dst, smdv_info.d_col, smdv_info.d_val, smdv_info.d_arr, 
        smdv_info.d_thread_start, num_rows, val_num
    );

    // Memcpy to Host
    SAFE_CALL( hipMemcpy(dst, smdv_info.d_dst, sizeof(float) * num_rows, hipMemcpyDeviceToHost) );
}


SmdvInfoOrdered init_mul (size_t num_rows, size_t num_cols, size_t val_num)
{
    SmdvInfoOrdered smdv_info;

    // Malloc
    SAFE_CALL( hipMalloc(&smdv_info.d_dst, sizeof(float) * num_rows) );
    SAFE_CALL( hipMalloc(&smdv_info.d_val, sizeof(float) * val_num) );
    SAFE_CALL( hipMalloc(&smdv_info.d_arr, sizeof(float) * num_cols) );
    SAFE_CALL( hipMalloc(&smdv_info.d_col, sizeof(size_t) * val_num) );
    SAFE_CALL( hipMalloc(&smdv_info.d_thread_start, sizeof(size_t) * (num_rows + 1)) );

    find_best_grid_linear(smdv_info.dim, num_rows);

    return smdv_info;
}


void deinit_mul (SmdvInfoOrdered &smdv_info)
{
    // Free
    SAFE_CALL( hipFree(smdv_info.d_dst) );
    SAFE_CALL( hipFree(smdv_info.d_val) );
    SAFE_CALL( hipFree(smdv_info.d_arr) );
    SAFE_CALL( hipFree(smdv_info.d_col) );
    SAFE_CALL( hipFree(smdv_info.d_thread_start) );
}


