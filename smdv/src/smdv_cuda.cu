#include "hip/hip_runtime.h"
#include "smdv_cuda.cuh"
#include "error.cuh"
#include "utils.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <malloc.h>
#include <math.h>

SmdvInfo init_mul (size_t num_rows, size_t num_cols, size_t val_num);
void deinit_mul (SmdvInfo smdv_info);

int main()
{
    size_t row[] = {0, 0, 1, 3, 4, 5, 5};
    size_t col[] = {1, 2, 3, 0, 1, 0, 2};
    float val[] = {1, 2, 5, 1, 3, 4, 2};
    
    size_t thread_start[] = {0, 2, 3, 3, 4, 5};

    float arr[] = {2, 6, 1, 3};

    float *res = (float *) malloc(sizeof(float) * 6);

    mul_cuda(res, col, val, arr, thread_start, 6, 4, 7);

    for (int i = 0; i < 6; i++) {
        printf("%.0lf ", res[i]);
    }

    printf("\n\n");

    return 0;
}

__global__
void mul_kernel (
    float *dst, size_t *col, float *val, float *arr, size_t *thread_start,
    size_t num_rows, size_t val_num
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread is in range
    if (idx >= num_rows)
        return;

    float p = 0;

    int thread_end = idx < (num_rows - 1) ? thread_start[idx + 1] : val_num;
    for (int i = thread_start[idx]; i < thread_end; i++)
    {
        p += val[i] * arr[col[i]];
    }

    dst[idx] = p;
}


void mul_cuda (
    float *dst, size_t *col, float *val, float *arr, size_t *thread_start,
    size_t num_rows, size_t num_cols, size_t val_num
)
{
    auto smdv_info = init_mul(num_rows, num_cols, val_num);

    // Memcpy
    SAFE_CALL( hipMemcpy(smdv_info.d_val, val, sizeof(float) * val_num, hipMemcpyHostToDevice) );
    SAFE_CALL( hipMemcpy(smdv_info.d_arr, arr, sizeof(float) * num_cols, hipMemcpyHostToDevice) );
    SAFE_CALL( hipMemcpy(smdv_info.d_col, col, sizeof(size_t) * val_num, hipMemcpyHostToDevice) );
    SAFE_CALL( hipMemcpy(smdv_info.d_thread_start, thread_start, sizeof(size_t) * num_rows, hipMemcpyHostToDevice) );

    dim3 dim_grid = dim3(smdv_info.dim.grid.x, smdv_info.dim.grid.y, smdv_info.dim.grid.z);
    dim3 dim_block = dim3(smdv_info.dim.block.x, smdv_info.dim.block.y, smdv_info.dim.block.z);

    // Kernel invocation
    mul_kernel <<< dim_grid, dim_block >>> (
        smdv_info.d_dst, smdv_info.d_col, smdv_info.d_val, smdv_info.d_arr, 
        smdv_info.d_thread_start, num_rows, val_num
    );

    // Memcpy to Host
    SAFE_CALL( hipMemcpy(dst, smdv_info.d_dst, sizeof(float) * num_rows, hipMemcpyDeviceToHost) );

    deinit_mul(smdv_info);
}


SmdvInfo init_mul (size_t num_rows, size_t num_cols, size_t val_num)
{
    SmdvInfo smdv_info;

    // Malloc
    SAFE_CALL( hipMalloc(&smdv_info.d_dst, sizeof(float) * num_rows) );
    SAFE_CALL( hipMalloc(&smdv_info.d_val, sizeof(float) * val_num) );
    SAFE_CALL( hipMalloc(&smdv_info.d_arr, sizeof(float) * num_cols) );
    SAFE_CALL( hipMalloc(&smdv_info.d_col, sizeof(size_t) * val_num) );
    SAFE_CALL( hipMalloc(&smdv_info.d_thread_start, sizeof(size_t) * num_rows) );

    find_best_grid_linear(smdv_info.dim, num_rows);

    return smdv_info;
}


void deinit_mul (SmdvInfo smdv_info)
{
    // Free
    SAFE_CALL( hipFree(smdv_info.d_dst) );
    SAFE_CALL( hipFree(smdv_info.d_val) );
    SAFE_CALL( hipFree(smdv_info.d_arr) );
    SAFE_CALL( hipFree(smdv_info.d_col) );
    SAFE_CALL( hipFree(smdv_info.d_thread_start) );
}


